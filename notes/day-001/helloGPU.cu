#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloFromGPU() {
    printf("Hello from the GPU!\n");
}

int main() {
    helloFromGPU<<<1, 1>>>();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Device sync failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Hello from the CPU!\n");
    return 0;
}

