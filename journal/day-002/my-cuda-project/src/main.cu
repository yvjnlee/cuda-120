#include <hip/hip_runtime.h>
#include "kernels.h"

int main() {
	dummyKernel<<<2, 4>>>();
	// hipDeviceSynchronize blocks CPU from accessing mem until GPU is finished its ops
	hipDeviceSynchronize();
	return 0;
}

