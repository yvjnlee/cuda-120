#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int count = 0;
    hipError_t err = hipGetDeviceCount(&count);

    if (err != hipSuccess) {
        printf("hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("CUDA device count: %d\n", count);

    for (int i = 0; i < count; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device %d: %s, compute capability %d.%d\n",
               i, prop.name, prop.major, prop.minor);
    }

    return 0;
}

